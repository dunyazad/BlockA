#include "hip/hip_runtime.h"
#include "CUDACommon.cuh"
#include <Neon/CUDA/CUDAMemoryPool.h>

namespace NeonCUDA
{
	// Hypothetical CUDA Mutex type
	typedef struct {
		int data;  // Data used for mutex implementation
	} cudaMutex_t;

	// Hypothetical CUDA Mutex functions
	__device__ void cudaLockMutex(cudaMutex_t* mutex) {
		printf("cudaLockMutex\n");

		printf("mutex->data : %d\n", mutex->data);

		// Implementation of locking mechanism
		// This is a simplified example and may not be suitable for production use
		while (atomicExch(&(mutex->data), 1) != 0) {
			// Spin until the lock is 
			//printf("In loop : %d\n", mutex->data);
		}

		printf("Locked : %d\n", mutex->data);
	}

	__device__ void cudaUnlockMutex(cudaMutex_t* mutex) {
		// Implementation of unlocking mechanism
		// This is a simplified example and may not be suitable for production use
		atomicExch(&(mutex->data), 0);
	}

	namespace MemoryPool
	{
		__device__ __managed__ char* allocatedMemory = nullptr;
		__device__ __managed__ char* currentMemory = nullptr;
		__device__ __managed__ size_t allocatedSize = 0;
		__device__ __managed__ cudaMutex_t* mutex = nullptr;

		__host__ bool Initialize(size_t size)
		{
			auto result = hipMalloc((void**)&allocatedMemory, size);
			CUDA_CHECK_ERROR(result);

			currentMemory = allocatedMemory;
			allocatedSize = size;

			result = hipMalloc((void**)&mutex, sizeof(cudaMutex_t));
			CUDA_CHECK_ERROR(result);

			printf("MemoryPool is successfully initialized.\n");

			return true;
		}

		__host__ bool Terminate()
		{
			auto result = hipFree(allocatedMemory);
			CUDA_CHECK_ERROR(result);

			printf("MemoryPool is successfully terminated.\n");

			return true;
		}

		template<typename T>
		__device__ T* New()
		{
			printf("New()\n");

			cudaLockMutex(mutex);

			printf("New() 11111111\n");

			auto output = currentMemory;
			currentMemory += sizeof(T);
			cudaUnlockMutex(mutex);
			return (T*)output;
		}
	}

	__global__ void Test()
	{
		int threadId = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadId < MemoryPool::allocatedSize / sizeof(int))
		{
			printf("[%d] ", threadId);

			auto value = MemoryPool::New<int>();

			printf("11111111\n");

			printf("value = %d\n", (int)value - (int)MemoryPool::allocatedMemory);
		}

		//printf("AllocatedSize: %d\n", MemoryPool::allocatedSize / sizeof(int));

		//for (size_t i = 0; i < MemoryPool::allocatedSize / sizeof(int); i++)
		//{
		//	auto value = MemoryPool::New<int>();

		//	//printf("[%d] value = %d\n", i, (int)value - (int)MemoryPool::allocatedMemory);
		//}
	}

	void MemoryPoolTest()
	{
		MemoryPool::Initialize(102400000);

		Test<<<1,10>>>();

		CUDA_CHECK_ERROR(hipGetLastError());

		MemoryPool::Terminate();
	}
}
