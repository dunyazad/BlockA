#include "hip/hip_runtime.h"
#include "CUDACommon.cuh"
#include <Neon/CUDA/CUDAList.h>

namespace NeonCUDA {

	struct ListElement
	{
		ListElement* previous = nullptr;
		ListElement* next = nullptr;
		size_t id = 0;
		size_t data;
		bool isInUse = false;
	};

	//__device__ size_t count = 0;

	class ListElementPool
	{
	public:
		ListElement* memory = nullptr;
		ListElement* availableBlock = nullptr;
		size_t totalAllocated = 0;
		size_t count = 0;

		bool Initialize(size_t size)
		{
			{
				hipError_t cudaStatus = hipMalloc((void**)&memory, size * sizeof(ListElement));

				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
					return false;
				}
			}

			{
				hipError_t cudaStatus = hipMemset((void*)memory, 0, size * sizeof(ListElement));

				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemset failed: %s\n", hipGetErrorString(cudaStatus));
					return false;
				}
			}

			availableBlock = memory;

			return true;
		}

		__device__ ListElement* New()
		{
			printf("ListElement* New()\n");

			auto result = availableBlock;
			count = atomicAdd(&count, 1);
			printf("11111111111\n");
			result->id = count;
			printf("22222222222\n");
			availableBlock += sizeof(ListElement);
			printf("33333333333\n");

			printf("count : %d\n", count);

			return result;
		}
	};

	__global__ void Fill(ListElement* memory, size_t size)
	{
		int threadId = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadId < size)
		{
			if (threadId > 0)
			{
				memory[threadId].previous = &memory[threadId - 1];
			}
			memory[threadId].data = threadId;

			if (threadId < size - 1)
			{
				memory[threadId].next = &memory[threadId + 1];
			}
		}
	}

	__global__ void FillFromDevice(ListElementPool* dev_pool, size_t size)
	{
		int threadId = blockIdx.x * blockDim.x + threadIdx.x;
		if (threadId < size)
		{
			printf("threadId = %d\n", threadId);
			
			auto element = dev_pool->New();
			//auto element = dev_pool.New();

			printf("element->id = %d\n", element->id);

			element->data = threadId+1;

			printf("element->data = %d\n", element->data);
		}
	}

	__global__ void Temp(ListElement* p, int* result)
	{
		for (size_t i = 0; i < 20; i++)
		{
			result[i] = p->data;
			p = p->next;
		}
	}

	void ListTestFunction()
	{
		const size_t size = 12000000;

		ListElementPool pool;
		pool.Initialize(size * 10);

		ListElementPool* dev_pool;
		hipMalloc(&dev_pool, sizeof(ListElementPool));
		hipMemcpy(dev_pool, &pool, sizeof(ListElementPool), hipMemcpyHostToDevice);

		//Fill<<<48000, 250>>>(pool.memory, size);
		FillFromDevice<<<1, 10>>>(dev_pool, size);

		//hipMemcpy(&pool, dev_pool, size * sizeof(ListElementPool), hipMemcpyDeviceToHost);

		hipMemcpy(&pool, dev_pool, sizeof(ListElementPool), hipMemcpyDeviceToHost);

		ListElement* host = new ListElement[size];
		hipMemcpy(host, pool.memory, size * sizeof(ListElement), hipMemcpyDeviceToHost);
		//for (size_t i = 0; i < 100; i++)
		//{
		//	printf("[%d].data : %d\n", i, host[i].id);
		//}

		//int* dev_result;
		//hipMalloc(&dev_result, sizeof(int) * 20);

		//Temp<<<1,1>>>(pool.memory, dev_result);

		//int* host_result = new int[20];
		//hipMemcpy(host_result, dev_result, sizeof(int) * 20, hipMemcpyDeviceToHost);

		//for (size_t i = 0; i < 20; i++)
		//{
		//	std::cout << "(" << i << ") " << host_result[i] << std::endl;
		//}



		//for (size_t i = 0; i < size; i++)
		//{
		//	std::cout << "host[" << i << "] data : " << host[i].previous << std::endl;
		//	if (20 == i)
		//	{
		//		break;
		//	}
		//}
	}

	template<typename T>
	class ObjectPool
	{
	public:
		ObjectPool()
		{
		}

		~ObjectPool()
		{
			Terminate();
		}

		void Initialize(size_t count)
		{
			if (false == initialized)
			{
				auto result = hipMalloc((void**)&objects, sizeof(T) * count);
				CUDA_CHECK_ERROR(result);

				current = objects;

				result = hipMalloc((void**)&device_pool, sizeof(ObjectPool<T>));
				CUDA_CHECK_ERROR(result);

				result = hipMemcpy(device_pool, this, sizeof(ObjectPool<T>), hipMemcpyDeviceToHost);

				printf("ObjectPool successfully initialized.\n");
			}
		}

		void Terminate()
		{
			if (false == terminated)
			{
				auto result = hipFree(objects);
				CUDA_CHECK_ERROR(result);

				printf("ObjectPool successfully terminated.\n");
			}
		}

		__device__ T* New()
		{
			if (0 == allocated)
			{
				current = objects;
			}

			current += sizeof(T);
			allocated++;
			return current - sizeof(T);
		}

		__host__ __device__ inline T* Objects() { return objects; }
		__host__ __device__ inline ObjectPool<T>* DevicePool() { return device_pool; }

	//private:
		ObjectPool<T>* device_pool = nullptr;

		T* objects = nullptr;
		T* current = nullptr;
		bool initialized = false;
		bool terminated = false;
		size_t allocated = 0;
	};

	/*__global__ void Alloc(ObjectPool<ListElement>* device_pool)
	{
		for (size_t i = 0; i < 100; i++)
		{
			printf("* %d\n", i);

			auto element = device_pool->New();

			printf("! %d\n", i);

			element->data = i + 1;

			CUDA_CHECK_ERROR(hipGetLastError());

			printf("- %d\n", i);

			printf("element->data : %d\n", element->data);
		}
	}

	void ObjectPoolTest()
	{
		ObjectPool<ListElement> pool;
		pool.Initialize(1024000);

		Alloc<<<1, 1>>>(pool.DevicePool());
		CUDA_CHECK_ERROR(hipGetLastError());

		ListElement* host = new ListElement[110];
		hipMemcpy(host, pool.Objects(), sizeof(ListElement) * 100, hipMemcpyDeviceToHost);

		for (size_t i = 0; i < 110; i++)
		{
			std::cout << "ListElement[" << i << "] : " << host[i].data << std::endl;
		}
	}*/

}
